#include <hip/hip_runtime.h>
#include <stdio.h>
#define THREAD_NUM 1024

typedef struct Histogram{
	int start_idx;
	int prefix_num;
} Hist;

__global__ void counting_sort_kernel(int arr[], int histarr[], int size)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(idx < size)
		atomicAdd(&histarr[arr[idx]], 1);
}

__global__ void output_kernel(int result[], Hist prefix[], int max_val)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
		
	__shared__ int count;
	count = prefix[idx].prefix_num;
	__shared__ int start_idx;
	start_idx = prefix[idx].start_idx;

	for(int i = 0; i< count; i++){
		result[start_idx + i] = idx;
	}

}

__host__ void counting_sort(int arr[], int size, int max_val)
{       
	int *d_arr;
	int *d_histogram;
	int *histogram = (int *)malloc(max_val * sizeof(int));

	hipMalloc(&d_arr, size *sizeof(int));
	hipMemcpy(d_arr, arr, size * sizeof(int), hipMemcpyHostToDevice); // initialize unsorted array

	hipMalloc(&d_histogram, max_val * sizeof(int));
	hipMemset(d_histogram, 0, max_val * sizeof(int)); // initialize histogram

	int block_num = size / THREAD_NUM;
	if(size % THREAD_NUM)
		block_num = block_num + 1; // Case for remainder;
	counting_sort_kernel <<< block_num, THREAD_NUM >>> (d_arr, d_histogram, size);

	hipMemcpy(histogram, d_histogram, max_val * sizeof(int), hipMemcpyDeviceToHost);
	//cudaFree(d_arr);
	hipFree(d_histogram);

	Hist *histo = (Hist *)malloc(max_val * sizeof(Hist));

	int sum = 0;
	for(int i = 0; i < max_val; i++){
		sum = sum + histogram[i];
		histo[i].prefix_num = histogram[i];
		histo[i].start_idx = sum - histogram[i];
	}

	//int *d_out;
	Hist *d_prefix;

	//cudaMalloc(&d_out, size * sizeof(int));
	hipMemset(d_arr, 0, size * sizeof(int));

	hipMalloc(&d_prefix, max_val * sizeof(Hist));
	hipMemcpy(d_prefix, histo, max_val *sizeof(Hist), hipMemcpyHostToDevice);

	block_num = max_val / THREAD_NUM;
	if(max_val % THREAD_NUM)
		block_num = block_num + 1;
	output_kernel <<< block_num, THREAD_NUM >>> (d_arr, d_prefix, max_val);
	
	hipMemcpy(arr, d_arr, size * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_arr);
	hipFree(d_prefix);

	free(histogram);
}


